#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include "glm/glm.hpp"
#include "kdtree.h"
#include ""
#include "common.h"

namespace ScanMatching {
	namespace GPU {

		__device__ void print_k(glm::vec3* points, int num) {
			for (int i = 0; i < num; i++) {
				printf("%f\t%f\t%f\n", points[i].x, points[i].y, points[i].z);
			}
		}

		__global__ void print_kernel(float* points, int num) {
			for (int i = 0; i < num; i++) {
				printf("%f\t%f\t%f\n", points[3 * i], points[3 * i + 1], points[3 * i + 2]);
			}
		}

		__global__ void print_v4_kernel(glm::vec4* points, int num) {
			for (int i = 0; i < num; i++) {
				printf("%f\t%f\t%f\n", points[i].x, points[i].y, points[i].z);
			}
		}

		__global__ void print_v3_kernel(glm::vec3* points, int num) {
			for (int i = 0; i < num; i++) {
				printf("%f\t%f\t%f\n", points[i].x, points[i].y, points[i].z);
			}
		}

		struct XComparator {
			__host__ __device__ inline bool operator() (const glm::vec3 a, const glm::vec3 b) {
				return a.x < b.x;
			}
		};
		struct YComparator {
			__host__ __device__ inline bool operator() (const glm::vec3 a, const glm::vec3 b) {
				return a.y < b.y;
			}
		};
		struct ZComparator {
			__host__ __device__ inline bool operator() (const glm::vec3 a, const glm::vec3 b) {
				return a.z < b.z;
			}
		};

		__device__ void buildTree(glm::vec4 *tree, glm::vec3 *points, int dim, int idx, int s, int e) {
			printf("In Tree: idx: %d dim: %d, s: %d e: %d\n", idx, dim, s, e);
			print_k(points, 2);
			if (s > e)
				return;
			if (dim == 0)
				thrust::sort(thrust::device, points + s, points + e, XComparator());
			if (dim == 1)
				thrust::sort(thrust::device, points + s, points + e, YComparator());
			if (dim == 2)
				thrust::sort(thrust::device, points + s, points + e, ZComparator());
			int mid = (s + e) / 2;
			tree[idx] = glm::vec4(points[mid].x, points[mid].y, points[mid].z, 1.0f);
			buildTree(tree, points, (dim + 1) % 3, 2 * idx + 1, s, mid - 1);
			buildTree(tree, points, (dim + 1) % 3, 2 * idx + 2, mid + 1, e);
			printf("Out Tree\n");
			print_k(points, 2);
		}

		__host__ void buildTreeH(glm::vec4 *tree, glm::vec3 *points, int dim, int idx, int s, int e) {
			if (s > e)
				return;
			if (dim == 0)
				thrust::sort(thrust::host, points + s, points + e, XComparator());
			if (dim == 1)
				thrust::sort(thrust::host, points + s, points + e, YComparator());
			if (dim == 2)
				thrust::sort(thrust::host, points + s, points + e, ZComparator());
			int mid = (s + e) / 2;
			tree[idx] = glm::vec4(points[mid].x, points[mid].y, points[mid].z, 1.0f);
			buildTreeH(tree, points, (dim + 1) % 3, 2 * idx + 1, s, mid - 1);
			buildTreeH(tree, points, (dim + 1) % 3, 2 * idx + 2, mid + 1, e);
		}

		__global__ void kernel_build_tree(glm::vec4* tree, glm::vec3* ypoints_vec, int ynum) {
			buildTree(tree, ypoints_vec, 0, 0, 0, ynum - 1);
		}

		// Builds KD Tree
		void build(glm::vec4 *tree, glm::vec3 *points, int xnum) {
			kernel_build_tree << <1, 1 >> > (tree, points, xnum);
		}

		// Builds KD Tree
		void buildHost(glm::vec4 *tree, glm::vec3 *points, int ynum, int size) {
			glm::vec4 *treeH = new glm::vec4[ynum];
			glm::vec3 *pointsH = new glm::vec3[ynum]; 
			hipMemcpy(pointsH, points, ynum * sizeof(glm::vec3), hipMemcpyDeviceToHost);
			buildTreeH(treeH, pointsH, 0, 0, 0, ynum - 1);
			hipMemcpy(tree, treeH, ynum * sizeof(glm::vec4), hipMemcpyHostToDevice);
		}
		__device__ void push(context* context_stack, int *top, context cur_context) {
			//printf("PUSH: %d, %d, %d\n", cur_context.idx, cur_context.dim, cur_context.good);
			context_stack[(*top)++] = cur_context;
			//printf("After PUSH: %d, %d, %d\n", context_stack[*idx - 1].idx, context_stack[*idx - 1].dim, context_stack[*idx - 1].good);
		}

		__device__ context pop(context* context_stack, int *top) {
			//printf("Before POP: %d, %d, %d\n", context_stack[*idx].idx, context_stack[*idx].dim, context_stack[*idx].good);
			return context_stack[(*top)--];
		}

		__device__ long potential_best_dist(glm::vec3 root_node, int dim, glm::vec3 query) {
			long dist;
			if (dim == 0) {
				dist = fabsf(query.x - root_node.x);
			}
			if (dim == 1) {
				dist = fabsf(query.y - root_node.y);
			}
			if (dim == 2) {
				dist = fabsf(query.z - root_node.z);
			}
			return dist;
		}

		__device__ void find_1NN(glm::vec4* tree, const glm::vec3 query, long nn_dist, glm::vec3* nn, const int size, context* context_stack) {
			int top = 0;
			context cur_context;
			cur_context.dim = 1;
			cur_context.good = true;
			cur_context.idx = 0;
			push(context_stack, &top, cur_context);
			while (top > 0) {
				context popped_context = pop(context_stack, &top);
				// Null Check
				if (tree[popped_context.idx].w >= 0.5) {
					glm::vec3 root_node = glm::vec3(tree[popped_context.idx].x, tree[popped_context.idx].y, tree[popped_context.idx].z);
					float dist = glm::distance(root_node, query);
					if (nn_dist > dist) {
						nn_dist = dist;
						(*nn).x = root_node.x;
						(*nn).y = root_node.y;
						(*nn).z = root_node.z;
					}
					// Prune
					if (!popped_context.good && potential_best_dist(root_node, popped_context.dim, query) > nn_dist) {
						continue;
					}
					// If Good Path or Bad Path with potential goodness

					// Good and Bad Side Calculation: Slightly Verbose
					int left = 2 * popped_context.idx + 1;
					int right = 2 * popped_context.idx + 2;
					int good_idx = -1;
					int bad_idx = -1;
					if (popped_context.dim == 0) {
						if (left < size && tree[left].w > 0.5) {
							if (tree[left].x <= query.x)
								good_idx = left;
							else
								bad_idx = left;
						}
						if (right < size && tree[right].w > 0.5) {
							if (tree[right].x <= query.x)
								good_idx = right;
							else
								bad_idx = right;
						}
					}
					if (popped_context.dim == 1) {
						if (left < size && tree[left].w > 0.5) {
							if (tree[left].y <= query.y)
								good_idx = left;
							else
								bad_idx = left;
						}
						if (right < size && tree[right].w > 0.5) {
							if (tree[right].y <= query.y)
								good_idx = right;
							else
								bad_idx = right;
						}
					}
					if (popped_context.dim == 2) {
						if (left < size && tree[left].w > 0.5) {
							if (tree[left].z <= query.z)
								good_idx = left;
							else
								bad_idx = left;
						}
						if (right < size && tree[right].w > 0.5) {
							if (tree[right].z <= query.z)
								good_idx = right;
							else
								bad_idx = right;
						}
					}
					if (bad_idx != -1) {
						//context bad_context = { bad_idx, false, (popped_context.dim + 1) % 3 };
						context bad_context;
						bad_context.dim = (popped_context.dim + 1) % 3;
						bad_context.good = false;
						bad_context.idx = bad_idx;
						push(context_stack, &top, bad_context);
					}
					if (good_idx != -1) {
						context good_context;
						cur_context.dim = (popped_context.dim + 1) % 3;
						cur_context.good = true;
						cur_context.idx = good_idx;
						//context good_context = { good_idx, true, (popped_context.dim + 1) % 3 };
						push(context_stack, &top, good_context);

					}
				}
			}
		}

		// Kernel to find correspondence of one point in xp and update in cyp
		__global__ void kernel_find_correspondences(glm::vec4* tree, float* xp, float* cyp, int xnum, int size, int height, context* stack) {
			int i = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (i < xnum) {
				glm::vec3 nn;
				find_1NN(tree, glm::vec3(xp[3 * i], xp[3 * i + 1], xp[3 * i + 2]), LONG_MAX, &nn, size, stack + i* height);
				cyp[3 * i] = nn.x;
				cyp[3 * i + 1] = nn.y;
				cyp[3 * i + 2] = nn.z;
			}
		}

		// Finds correspondences for all points in xp from the KD Tree: tree
		void find_correspondences(float* xp, glm::vec4* tree, float* cyp, int xnum, int ynum, int blockSize, context* stack){
			dim3 xnumBlocks((xnum + blockSize - 1) / blockSize);
			int size = 1 << ilog2ceil(ynum);
			kernel_find_correspondences << <xnumBlocks, blockSize >> > (tree, xp, cyp, xnum, size, ilog2ceil(ynum), stack);
		}
	}
}
