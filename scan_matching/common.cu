#include "common.h"
#include ""

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
	exit(EXIT_FAILURE);
}

void printArray(int n, float *a, bool abridged = false) {
	printf("    [ ");
	for (int i = 0; i < n; i++) {
		if (abridged && i + 2 == 15 && n > 16) {
			i = n - 2;
			printf("... ");
		}
		printf("%f ", a[i]);
	}
	printf("]\n");
}

void printCudaArray(int size, float* data) {
	float *d_data = new float[size];
	hipMemcpy(d_data, data, size * sizeof(float), hipMemcpyDeviceToHost);
	printArray(size, d_data, true);
}

void printCuda2DArray(int height, int width, float* data) {
	float *d_data = new float[width*height];
	hipMemcpy(d_data, data, width*height * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < height; i++)
		printArray(width, d_data + i * width, true);
}
